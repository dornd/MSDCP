#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define INF INT_MAX

using namespace std;

int N;

template <typename T>
struct Edge {
    int u, v;
    T dissimilarity;
};

template <typename T>
struct edgeCmp {
    __host__ __device__
    bool operator()(const Edge<T>& e1, const Edge<T>& e2) const {
        return e1.dissimilarity < e2.dissimilarity;
    }
};

struct Threads {
    dim3 dimBlocks;
    dim3 dimGrids;

    Threads(int x, int y) {

        if (x > 32) {
            dimBlocks.x = (int)ceil(x/32.0);
            dimGrids.x = 32;
        } else {
            dimGrids.x = x;
        }

        if (y > 32) {
            dimBlocks.y = (int)ceil(y/32.0);
            dimGrids.y = 32;
        } else {
            dimGrids.y = y;
        }

    }
};

int findSet(int* parent, int i) {
    return (parent[i] == i) ? i : (parent[i] = findSet(parent, parent[i]));
}

template <typename T>
thrust::host_vector<int> getCandidateDiameter(thrust::device_vector<Edge<T>> edges) {
    
    thrust::host_vector<int> eT, eC;
    thrust::host_vector<thrust::host_vector<int>> MST(N);
    int* parent = new int[N];
    int* rank = new int[N];

    for (int i = 0; i < N; ++i)
        parent[i] = i, rank[i] = 0;
    
    for (int i = edges.size()-1; i >= 0; i--) {
        Edge<T> e = edges[i];

        int u = e.u;
        int v = e.v;
        
        int x = findSet(parent, u);
        int y = findSet(parent, v);

        if (x != y) {

            eT.push_back(i);
            if (rank[x] > rank[y]) {
                parent[y] = x;
            } else {
                parent[x] = y;
                if (rank[x] == rank[y])
                    rank[y]++;
            }

            MST[u].push_back(v);
            MST[v].push_back(u);

        } else {
            eC.push_back(i);
        }
    }

    thrust::host_vector<int> dist(N, INF);
    dist[0] = 0;

    queue<int> q;
    q.push(0);

    while (!q.empty()) {
        int u = q.front();
        q.pop();

        for (int j = 0; j < (int)MST[u].size(); ++j) {
            int v = MST[u][j];
            if (dist[v] == INT_MAX) {
                dist[v] = dist[u]+1;
                q.push(v);
            }
        }
    }

    int b, r;
    for (int i = 0; i < (int)eC.size(); ++i) {
        Edge<T> e = edges[eC[i]];

        int u = e.u;
        int v = e.v;
        b = e.dissimilarity;

        if (dist[u] % 2 == dist[v] % 2) {
            r = eC[i]; 
            break;
        }
    }

    thrust::host_vector<int> eB;
    for (int i = 0; i < (int)eT.size(); ++i) {

        Edge<T> e = edges[eT[i]];
        int w = e.dissimilarity;

        if (w > b)
            eB.push_back(eT[i]);
        else
            break;
    }
    
    eB.push_back(r);

    return eB;
}

template<typename T>
__global__ void construct_boolean_expression(
            bool* d_G, Edge<T>* E, T d1, T d2, int V, int N, int len_E) {

    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = i*N+j;

    if (idx < len_E) {

        int u_i = E[idx].u;
        int u_j = E[idx].v;
        int d_ij = E[idx].dissimilarity;

        if (d1 < d_ij) {

            d_G[u_i*V+(u_j+N)] = true;  // (u_i' -> u_j)
            d_G[u_j*V+(u_i+N)] = true;  // (u_j' -> u_i)
            d_G[(u_i+N)*V+u_j] = true;  // (u_i  -> u_j')
            d_G[(u_j+N)*V+u_i] = true;  // (u_j  -> u_i')

        } else if (d2 < d_ij) {

            d_G[(u_i+N)*V+u_j] = true;  // (u_i -> u_j')
            d_G[(u_j+N)*V+u_i] = true;  // (u_j -> u_i')

        }
    }
}

__global__ 
void initDiagonal(bool* d_G, int V) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < V)
        d_G[i*V+i] = true;
}

__global__
void mmul(bool* A, bool* B, int V) {
    int i = blockDim.y * blockIdx.y + threadIdx.y; 
    int j = blockDim.x * blockIdx.x + threadIdx.x; 

    if (i < V && j < V) {
            
        for (int k = 0; k < V; ++k)
            B[i*V+j] = B[i*V+j] || (A[i*V+k] && A[k*V+j]);
    }
}

void square(bool *A, int V, Threads t) {

    bool* B;

    hipMallocManaged(&B, V*V*sizeof(bool)); 

    mmul<<<t.dimBlocks, t.dimGrids>>>(A, B, V);
    hipDeviceSynchronize();

    hipMemcpy(A, B, V*V*sizeof(bool), hipMemcpyDeviceToDevice);
    hipFree(B);
}

__global__
void check_cycle(bool* d_G, bool* cycle, int V, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N) {

        cycle[i] = true; // no cycle = true, cycle = false
        __syncthreads();

        cycle[i] = !(d_G[i*V+(i+N)] && d_G[(i+N)*V+i]);
    }

}

bool check_satisfy(bool* d_G, int V) {

    struct Threads tv = Threads(V, 1);
    struct Threads tn = Threads(N, 1);
    struct Threads t = Threads(V, V);

    initDiagonal<<<tv.dimBlocks, tv.dimGrids>>>(d_G, V);
    hipDeviceSynchronize();

    for(int k = 1; k <= V; k <<= 1)
        square(d_G, V, t);
    square(d_G, V, t);

    bool* d_cycle;
    bool* cycle = new bool[N];
    hipMalloc(&d_cycle, N*sizeof(bool));

    check_cycle<<<tn.dimBlocks, tn.dimGrids>>>(d_G, d_cycle, V, N);
    hipDeviceSynchronize();

    hipMemcpy(cycle, d_cycle, N*sizeof(bool), hipMemcpyDeviceToHost);

    bool satisfy = thrust::reduce(cycle, cycle+N, true, thrust::bit_and<bool>());

    return satisfy;
}

template <typename T>
T solve(thrust::device_vector<Edge<T>> edges) {

    int V = 2*N;
    thrust::sort(edges.begin(), edges.end(), edgeCmp<T>());
    thrust::host_vector<int> eB = getCandidateDiameter<int>(edges);

    Edge<T> e_ans = edges[edges.size()-1];
    T ans = e_ans.dissimilarity;

    struct Threads t_n2 = Threads(N, N);
    struct Threads t_v2 = Threads(V, V);
    Edge<T>* e_ptr = thrust::raw_pointer_cast(&edges[0]);

    for (int i = 0; i < (int)eB.size(); ++i) {
        int l = 0;
        int r = eB[i]-1;

        Edge<T> e1 = edges[eB[i]];
        T d1 = e1.dissimilarity;

        while (l <= r) {
            int m = (l+r)/2;
            Edge<T> e2 = edges[m];
            T d2 = e2.dissimilarity;
            
            bool* d_G, *cycle;
            hipMallocManaged(&d_G, V*V*sizeof(bool));
            hipMallocManaged(&cycle, N*sizeof(bool));

            construct_boolean_expression<T><<<t_n2.dimBlocks, t_n2.dimGrids>>>(
                                        d_G, e_ptr, d1, d2, V, N, (int)edges.size());
            hipDeviceSynchronize();

            bool satisfy = check_satisfy(d_G, V);

            if (satisfy)
                r = m-1, ans = min(ans, d1+d2);
            else 
                l = m+1;

            hipFree(d_G);
            hipFree(cycle);

        }
    }

    return ans;
}

int main() {

    while (cin >> N) {
        
        thrust::device_vector<Edge<int>> edges;
        for (int i = 0; i < N; i++) {
            for (int j = i+1; j < N; j++) {
                int d;
                cin >> d;
                Edge<int> e = {i, j, d};
                edges.push_back(e);
            }

            Edge<int> e = {i, i, 0};
            edges.push_back(e);
        }
        
        clock_t begin = clock();

        int ans = solve<int>(edges);

        clock_t end = clock();
        double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
        printf("GPU time(secs) = %.10lf\n", elapsed_secs);

        printf("%d\n", ans);
    }
    
    return 0;
}
